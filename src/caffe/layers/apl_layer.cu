#include "hip/hip_runtime.h"

// Copyright 2014 BVLC and contributors.

#include <hipblas.h>

#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
	__global__ void APLForwardSum(const int n, int s, const Dtype* in, Dtype* out, const Dtype* neuron_weight, const Dtype* neuron_offset, Dtype* maxs_data, int sums_, int K_) {
		CUDA_KERNEL_LOOP(index, n) {
			int exPos = ((int) index / K_) * K_;
			int exPosSums = exPos*sums_;
			int k = index % K_;
			int sumPos = k*sums_;

			if (s == 0) {
				out[index] = in[index] > 0 ? in[index] : 0;
			}
			maxs_data[exPosSums + sumPos + s] = max(-in[index] + neuron_offset[sumPos + s], Dtype(0));
			out[index] += neuron_weight[sumPos + s]*maxs_data[exPosSums + sumPos + s];
		}
	}

template <typename Dtype>
	__global__ void APLForwardSumHardcode(const int n, const Dtype* in, Dtype* out, const Dtype* neuron_weight, const Dtype* neuron_offset, Dtype* maxs_data, int sums_, int K_) {
		CUDA_KERNEL_LOOP(index, n) {
			int exPos = ((int) index / K_) * K_;
			int exPosSums = exPos*sums_;
			int k = index % K_;
			int sumPos = k*sums_;

			switch (sums_) {
				case 1 : { 
									 maxs_data[exPosSums + sumPos + 0] = max(-in[index] + neuron_offset[sumPos + 0], Dtype(0));

									 Dtype inMax = in[index] > 0 ? in[index] : 0;
									 out[index] = inMax +  neuron_weight[sumPos + 0]*maxs_data[exPosSums + sumPos + 0];
									 break;
								 }
				case 2 : {
									 maxs_data[exPosSums + sumPos + 0] = max(-in[index] + neuron_offset[sumPos + 0], Dtype(0));
									 maxs_data[exPosSums + sumPos + 1] = max(-in[index] + neuron_offset[sumPos + 1], Dtype(0));

									 Dtype inMax = in[index] > 0 ? in[index] : 0;
									 out[index] = inMax +  neuron_weight[sumPos + 0]*maxs_data[exPosSums + sumPos + 0] + neuron_weight[sumPos + 1]*maxs_data[exPosSums + sumPos + 1];
									 break;
								 }
				case 3 : {
									 maxs_data[exPosSums + sumPos + 0] = max(-in[index] + neuron_offset[sumPos + 0], Dtype(0));
									 maxs_data[exPosSums + sumPos + 1] = max(-in[index] + neuron_offset[sumPos + 1], Dtype(0));
									 maxs_data[exPosSums + sumPos + 2] = max(-in[index] + neuron_offset[sumPos + 2], Dtype(0));

									 Dtype inMax = in[index] > 0 ? in[index] : 0;
									 out[index] = inMax +  neuron_weight[sumPos + 0]*maxs_data[exPosSums + sumPos + 0] + neuron_weight[sumPos + 1]*maxs_data[exPosSums + sumPos + 1] + neuron_weight[sumPos + 2]*maxs_data[exPosSums + sumPos + 2];
									 break;
								 }
				case 4 : {
									 maxs_data[exPosSums + sumPos + 0] = max(-in[index] + neuron_offset[sumPos + 0], Dtype(0));
									 maxs_data[exPosSums + sumPos + 1] = max(-in[index] + neuron_offset[sumPos + 1], Dtype(0));
									 maxs_data[exPosSums + sumPos + 2] = max(-in[index] + neuron_offset[sumPos + 2], Dtype(0));
									 maxs_data[exPosSums + sumPos + 3] = max(-in[index] + neuron_offset[sumPos + 3], Dtype(0));

									 Dtype inMax = in[index] > 0 ? in[index] : 0;
									 out[index] = inMax +  neuron_weight[sumPos + 0]*maxs_data[exPosSums + sumPos + 0] + neuron_weight[sumPos + 1]*maxs_data[exPosSums + sumPos + 1] + neuron_weight[sumPos + 2]*maxs_data[exPosSums + sumPos + 2] + neuron_weight[sumPos + 3]*maxs_data[exPosSums + sumPos + 3];
									 break;
								 }
				case 5 : {
									 maxs_data[exPosSums + sumPos + 0] = max(-in[index] + neuron_offset[sumPos + 0], Dtype(0));
									 maxs_data[exPosSums + sumPos + 1] = max(-in[index] + neuron_offset[sumPos + 1], Dtype(0));
									 maxs_data[exPosSums + sumPos + 2] = max(-in[index] + neuron_offset[sumPos + 2], Dtype(0));
									 maxs_data[exPosSums + sumPos + 3] = max(-in[index] + neuron_offset[sumPos + 3], Dtype(0));
									 maxs_data[exPosSums + sumPos + 4] = max(-in[index] + neuron_offset[sumPos + 4], Dtype(0));

									 Dtype inMax = in[index] > 0 ? in[index] : 0;
									 out[index] = inMax +  neuron_weight[sumPos + 0]*maxs_data[exPosSums + sumPos + 0] + neuron_weight[sumPos + 1]*maxs_data[exPosSums + sumPos + 1] + neuron_weight[sumPos + 2]*maxs_data[exPosSums + sumPos + 2] + neuron_weight[sumPos + 3]*maxs_data[exPosSums + sumPos + 3] + neuron_weight[sumPos + 4]*maxs_data[exPosSums + sumPos + 4];
									 break;
								 }
			}
		}
	}

template <typename Dtype>
	void APLLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
			const vector<Blob<Dtype>*>& top) {
		//Forward_cpu(bottom,top);

		//Initialize